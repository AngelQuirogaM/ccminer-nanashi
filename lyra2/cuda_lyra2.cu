#include "hip/hip_runtime.h"
/**
* Lyra2 (v1) cuda implementation based on djm34 work - SM 5/5.2
* tpruvot@github 2015
*/

#include <stdio.h>
#include <memory.h>

#define TPB52 32
#define TPB50 16
#define TPB30 16
#define TPB20 16

#ifdef __INTELLISENSE__
/* just for vstudio code colors */
#define __CUDA_ARCH__ 500
#endif

#include "cuda_lyra2_vectors.h"

#ifdef __INTELLISENSE__
/* just for vstudio code colors */
#if __CUDA_ARCH__ >= 300
__device__ uint32_t __shfl(uint32_t a, uint32_t b, uint32_t c);
#endif
#endif

#define Nrow 8
#define Ncol 8
#define memshift 3

__device__ uint2x4 *DState;

__device__ __forceinline__ uint2 LD4S(const int index)
{
	extern __shared__ uint2 shared_mem[];

	return shared_mem[(index * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x];
}

__device__ __forceinline__ void ST4S(const int index, const uint2 data)
{
	extern __shared__ uint2 shared_mem[];

	shared_mem[(index * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x] = data;
}

#if __CUDA_ARCH__ >= 300
__device__ __forceinline__ uint32_t WarpShuffle(uint32_t a, uint32_t b, uint32_t c)
{
	return __shfl(a, b, c);
}

__device__ __forceinline__ uint2 WarpShuffle(uint2 a, uint32_t b, uint32_t c)
{
	return make_uint2(__shfl(a.x, b, c), __shfl(a.y, b, c));
}

__device__ __forceinline__ void WarpShuffle3(uint2 &a1, uint2 &a2, uint2 &a3, uint32_t b1, uint32_t b2, uint32_t b3, uint32_t c)
{
	a1 = WarpShuffle(a1, b1, c);
	a2 = WarpShuffle(a2, b2, c);
	a3 = WarpShuffle(a3, b3, c);
}

#else
__device__ __forceinline__ uint32_t WarpShuffle(uint32_t a, uint32_t b, uint32_t c)
{
	extern __shared__ uint2 shared_mem[];

	const uint32_t thread = blockDim.x * threadIdx.y + threadIdx.x;
	uint32_t *_ptr = (uint32_t*)shared_mem;

	__threadfence_block();
	uint32_t buf = _ptr[thread];

	_ptr[thread] = a;
	__threadfence_block();
	uint32_t result = _ptr[(thread&~(c - 1)) + (b&(c - 1))];

	__threadfence_block();
	_ptr[thread] = buf;

	__threadfence_block();
	return result;
}

__device__ __forceinline__ uint2 WarpShuffle(uint2 a, uint32_t b, uint32_t c)
{
	extern __shared__ uint2 shared_mem[];

	const uint32_t thread = blockDim.x * threadIdx.y + threadIdx.x;

	__threadfence_block();
	uint2 buf = shared_mem[thread];

	shared_mem[thread] = a;
	__threadfence_block();
	uint2 result = shared_mem[(thread&~(c - 1)) + (b&(c - 1))];

	__threadfence_block();
	shared_mem[thread] = buf;

	__threadfence_block();
	return result;
}

__device__ __forceinline__ void WarpShuffle3(uint2 &a1, uint2 &a2, uint2 &a3, uint32_t b1, uint32_t b2, uint32_t b3, uint32_t c)
{
	extern __shared__ uint2 shared_mem[];

	const uint32_t thread = blockDim.x * threadIdx.y + threadIdx.x;

	__threadfence_block();
	uint2 buf = shared_mem[thread];

	shared_mem[thread] = a1;
	__threadfence_block();
	a1 = shared_mem[(thread&~(c - 1)) + (b1&(c - 1))];
	__threadfence_block();
	shared_mem[thread] = a2;
	__threadfence_block();
	a2 = shared_mem[(thread&~(c - 1)) + (b2&(c - 1))];
	__threadfence_block();
	shared_mem[thread] = a3;
	__threadfence_block();
	a3 = shared_mem[(thread&~(c - 1)) + (b3&(c - 1))];

	__threadfence_block();
	shared_mem[thread] = buf;
	__threadfence_block();
}

#endif

static __device__ __forceinline__
void Gfunc(uint2 &a, uint2 &b, uint2 &c, uint2 &d)
{
	a += b; d = eorswap32(a, d);
	c += d; b ^= c; b = ROR24(b);
	a += b; d ^= a; d = ROR16(d);
	c += d; b ^= c; b = ROR2(b, 63);
}

__device__ __forceinline__ void round_lyra(uint2 s[4])
{
	Gfunc(s[0], s[1], s[2], s[3]);
	WarpShuffle3(s[1], s[2], s[3], threadIdx.x + 1, threadIdx.x + 2, threadIdx.x + 3, 4);
	Gfunc(s[0], s[1], s[2], s[3]);
	WarpShuffle3(s[1], s[2], s[3], threadIdx.x + 3, threadIdx.x + 2, threadIdx.x + 1, 4);
}

static __device__ __forceinline__
void round_lyra(uint2x4* s)
{
	Gfunc(s[0].x, s[1].x, s[2].x, s[3].x);
	Gfunc(s[0].y, s[1].y, s[2].y, s[3].y);
	Gfunc(s[0].z, s[1].z, s[2].z, s[3].z);
	Gfunc(s[0].w, s[1].w, s[2].w, s[3].w);
	Gfunc(s[0].x, s[1].y, s[2].z, s[3].w);
	Gfunc(s[0].y, s[1].z, s[2].w, s[3].x);
	Gfunc(s[0].z, s[1].w, s[2].x, s[3].y);
	Gfunc(s[0].w, s[1].x, s[2].y, s[3].z);
}

static __device__ __forceinline__
void reduceDuplex(uint2 state[4], uint32_t thread)
{
	uint2 state1[3];

#if __CUDA_ARCH__ > 500
#pragma unroll
#endif
	for (int i = 0; i < Nrow; i++)
	{
		const uint32_t s0 = memshift * Ncol * 0 + memshift * (Ncol - 1) - i * memshift;
#pragma unroll
		for (int j = 0; j < 3; j++)
			ST4S(s0 + j, state[j]);

		round_lyra(state);
	}

#pragma unroll 4
	for (int i = 0; i < Nrow; i++)
	{
		const uint32_t s0 = memshift * Ncol * 0 + i * memshift;
		const uint32_t s1 = memshift * Ncol * 1 + memshift * (Ncol - 1) - i*memshift;

		for (int j = 0; j < 3; j++)
			state1[j] = LD4S(s0 + j);
		for (int j = 0; j < 3; j++)
			state[j] ^= state1[j];

		round_lyra(state);

		for (int j = 0; j < 3; j++)
			state1[j] ^= state[j];
		for (int j = 0; j < 3; j++)
			ST4S(s1 + j, state1[j]);
	}
}

static __device__ __forceinline__
void reduceDuplexRowSetup(const int rowIn, const int rowInOut, const int rowOut, uint2 state[4], uint32_t thread)
{
	uint2 state1[3], state2[3];

#pragma unroll 1
	for (int i = 0; i < Nrow; i++)
	{
		const uint32_t s1 = memshift * Ncol * rowIn + i * memshift;
		const uint32_t s2 = memshift * Ncol * rowInOut + i * memshift;
		const uint32_t s3 = memshift * Ncol * rowOut + memshift * (Ncol - 1) - i*memshift;

		for (int j = 0; j < 3; j++)
			state1[j] = LD4S(s1 + j);
		for (int j = 0; j < 3; j++)
			state2[j] = LD4S(s2 + j);
		for (int j = 0; j < 3; j++)
			state[j] ^= state1[j] + state2[j];

		round_lyra(state);

		for (int j = 0; j < 3; j++)
			ST4S(s3 + j, state1[j] ^ state[j]);

		//���O�̃X���b�h����f�[�^��Ⴄ(�����Ɉ��̃X���b�h�Ƀf�[�^�𑗂�)
		uint2 Data0 = state[0];
		uint2 Data1 = state[1];
		uint2 Data2 = state[2];
		WarpShuffle3(Data0, Data1, Data2, threadIdx.x - 1, threadIdx.x - 1, threadIdx.x - 1, 4);

		if (threadIdx.x == 0)
		{
			state2[0] ^= Data2;
			state2[1] ^= Data0;
			state2[2] ^= Data1;
		}
		else
		{
			state2[0] ^= Data0;
			state2[1] ^= Data1;
			state2[2] ^= Data2;
		}

		for (int j = 0; j < 3; j++)
			ST4S(s2 + j, state2[j]);
	}
}

static __device__ __forceinline__
void reduceDuplexRowt(const int rowIn, const int rowInOut, const int rowOut, uint2 state[4], const uint32_t thread)
{
	for (int i = 0; i < Nrow; i++)
	{
		uint2 state1[3], state2[3];

		const uint32_t s1 = memshift * Ncol * rowIn + i * memshift;
		const uint32_t s2 = memshift * Ncol * rowInOut + i * memshift;
		const uint32_t s3 = memshift * Ncol * rowOut + i * memshift;

#pragma unroll
		for (int j = 0; j < 3; j++) {
			state1[j] = LD4S(s1 + j);
			state2[j] = LD4S(s2 + j);
		}

#pragma unroll
		for (int j = 0; j < 3; j++)
			state[j] ^= state1[j] + state2[j];

		round_lyra(state);

		//���O�̃X���b�h����f�[�^��Ⴄ(�����Ɉ��̃X���b�h�Ƀf�[�^�𑗂�)
		uint2 Data0 = state[0];
		uint2 Data1 = state[1];
		uint2 Data2 = state[2];
		WarpShuffle3(Data0, Data1, Data2, threadIdx.x - 1, threadIdx.x - 1, threadIdx.x - 1, 4);

		if (threadIdx.x == 0)
		{
			state2[0] ^= Data2;
			state2[1] ^= Data0;
			state2[2] ^= Data1;
		}
		else
		{
			state2[0] ^= Data0;
			state2[1] ^= Data1;
			state2[2] ^= Data2;
		}

#pragma unroll
		for (int j = 0; j < 3; j++)
			ST4S(s2 + j, state2[j]);
#pragma unroll
		for (int j = 0; j < 3; j++)
			ST4S(s3 + j, LD4S(s3 + j) ^ state[j]);
	}
}

static __device__ __forceinline__
void reduceDuplexRowt_8(const int rowInOut, uint2* state, const uint32_t thread)
{

	uint2 state1[3], state2[3], last[3];

	const uint32_t ps1 = memshift * Ncol * 2;
	const uint32_t ps2 = memshift * Ncol * rowInOut;
	const uint32_t ps3 = memshift * Ncol * 5;

#pragma unroll
	for (int j = 0; j < 3; j++)
		last[j] = LD4S(ps2 + j);

#pragma unroll 
	for (int j = 0; j < 3; j++)
		state[j] ^= LD4S(ps1 + j) + last[j];

	round_lyra(state);

	//���O�̃X���b�h����f�[�^��Ⴄ(�����Ɉ��̃X���b�h�Ƀf�[�^�𑗂�)
	uint2 Data0 = state[0];
	uint2 Data1 = state[1];
	uint2 Data2 = state[2];
	WarpShuffle3(Data0, Data1, Data2, threadIdx.x - 1, threadIdx.x - 1, threadIdx.x - 1, 4);

	if (threadIdx.x == 0)
	{
		last[0] ^= Data2;
		last[1] ^= Data0;
		last[2] ^= Data1;
	}
	else
	{
		last[0] ^= Data0;
		last[1] ^= Data1;
		last[2] ^= Data2;
	}

	if (rowInOut == 5)
	{
#pragma unroll 
		for (int j = 0; j < 3; j++)
			last[j] ^= state[j];
	}

	for (int i = 1; i < Nrow; i++)
	{

		const uint32_t s1 = memshift * Ncol * 2 + i * memshift;
		const uint32_t s2 = memshift * Ncol * rowInOut + i * memshift;

#pragma unroll 
		for (int j = 0; j < 3; j++)
			state[j] ^= LD4S(s1 + j) + LD4S(s2 + j);

		round_lyra(state);
	}


#pragma unroll
	for (int j = 0; j < 3; j++)
		state[j] ^= last[j];
}

__constant__ uint28 blake2b_IV[2] = {
	0xf3bcc908lu, 0x6a09e667lu,
	0x84caa73blu, 0xbb67ae85lu,
	0xfe94f82blu, 0x3c6ef372lu,
	0x5f1d36f1lu, 0xa54ff53alu,
	0xade682d1lu, 0x510e527flu,
	0x2b3e6c1flu, 0x9b05688clu,
	0xfb41bd6blu, 0x1f83d9ablu,
	0x137e2179lu, 0x5be0cd19lu
};

__global__ __launch_bounds__(64, 1)
void lyra2_gpu_hash_32_1(uint32_t threads, uint32_t startNounce, uint2 *g_hash)
{
	const uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);

	if (thread < threads)
	{
		uint2x4 state[4];

		state[0].x = state[1].x = __ldg(&g_hash[thread + threads * 0]);
		state[0].y = state[1].y = __ldg(&g_hash[thread + threads * 1]);
		state[0].z = state[1].z = __ldg(&g_hash[thread + threads * 2]);
		state[0].w = state[1].w = __ldg(&g_hash[thread + threads * 3]);
		state[2] = blake2b_IV[0];
		state[3] = blake2b_IV[1];

		for (int i = 0; i<24; i++)
			round_lyra(state); //because 12 is not enough

		DState[blockDim.x * gridDim.x * 0 + blockDim.x * blockIdx.x + threadIdx.x] = state[0];
		DState[blockDim.x * gridDim.x * 1 + blockDim.x * blockIdx.x + threadIdx.x] = state[1];
		DState[blockDim.x * gridDim.x * 2 + blockDim.x * blockIdx.x + threadIdx.x] = state[2];
		DState[blockDim.x * gridDim.x * 3 + blockDim.x * blockIdx.x + threadIdx.x] = state[3];
	}
}

#if __CUDA_ARCH__ < 300
__global__ __launch_bounds__(TPB20, 1)
#elif __CUDA_ARCH__ < 500
__global__ __launch_bounds__(TPB30, 1)
#elif __CUDA_ARCH__ == 500
__global__ __launch_bounds__(TPB50, 1)
#else
__global__ __launch_bounds__(TPB52, 1)
#endif
void lyra2_gpu_hash_32_2(uint32_t threads, uint32_t startNounce, uint64_t *g_hash)
{
	const uint32_t thread = blockDim.y * blockIdx.x + threadIdx.y;

	if (thread < threads)
	{
		uint2 state[4];
		state[0] = ((uint2*)DState)[(0 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x];
		state[1] = ((uint2*)DState)[(1 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x];
		state[2] = ((uint2*)DState)[(2 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x];
		state[3] = ((uint2*)DState)[(3 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x];

		reduceDuplex(state, thread);

		reduceDuplexRowSetup(1, 0, 2, state, thread);
		reduceDuplexRowSetup(2, 1, 3, state, thread);
		reduceDuplexRowSetup(3, 0, 4, state, thread);
		reduceDuplexRowSetup(4, 3, 5, state, thread);
		reduceDuplexRowSetup(5, 2, 6, state, thread);
		reduceDuplexRowSetup(6, 1, 7, state, thread);

		uint32_t rowa = WarpShuffle(state[0].x, 0, 4) & 7;
		reduceDuplexRowt(7, rowa, 0, state, thread);
		rowa = WarpShuffle(state[0].x, 0, 4) & 7;
		reduceDuplexRowt(0, rowa, 3, state, thread);
		rowa = WarpShuffle(state[0].x, 0, 4) & 7;
		reduceDuplexRowt(3, rowa, 6, state, thread);
		rowa = WarpShuffle(state[0].x, 0, 4) & 7;
		reduceDuplexRowt(6, rowa, 1, state, thread);
		rowa = WarpShuffle(state[0].x, 0, 4) & 7;
		reduceDuplexRowt(1, rowa, 4, state, thread);
		rowa = WarpShuffle(state[0].x, 0, 4) & 7;
		reduceDuplexRowt(4, rowa, 7, state, thread);
		rowa = WarpShuffle(state[0].x, 0, 4) & 7;
		reduceDuplexRowt(7, rowa, 2, state, thread);
		rowa = WarpShuffle(state[0].x, 0, 4) & 7;
		reduceDuplexRowt_8(rowa, state, thread);

		((uint2*)DState)[(0 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x] = state[0];
		((uint2*)DState)[(1 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x] = state[1];
		((uint2*)DState)[(2 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x] = state[2];
		((uint2*)DState)[(3 * gridDim.x * blockDim.y + thread) * blockDim.x + threadIdx.x] = state[3];
	}
}

__global__ __launch_bounds__(64, 1)
void lyra2_gpu_hash_32_3(uint32_t threads, uint32_t startNounce, uint2 *g_hash)
{
	const uint32_t thread = blockDim.x * blockIdx.x + threadIdx.x;

	uint28 state[4];

	if (thread < threads)
	{
		state[0] = __ldg4(&DState[blockDim.x * gridDim.x * 0 + blockDim.x * blockIdx.x + threadIdx.x]);
		state[1] = __ldg4(&DState[blockDim.x * gridDim.x * 1 + blockDim.x * blockIdx.x + threadIdx.x]);
		state[2] = __ldg4(&DState[blockDim.x * gridDim.x * 2 + blockDim.x * blockIdx.x + threadIdx.x]);
		state[3] = __ldg4(&DState[blockDim.x * gridDim.x * 3 + blockDim.x * blockIdx.x + threadIdx.x]);

		for (int i = 0; i < 12; i++)
			round_lyra(state);

		g_hash[thread + threads * 0] = state[0].x;
		g_hash[thread + threads * 1] = state[0].y;
		g_hash[thread + threads * 2] = state[0].z;
		g_hash[thread + threads * 3] = state[0].w;

	} //thread
}

__host__
void lyra2_cpu_init(int thr_id, uint32_t threads, uint64_t *d_matrix)
{
	int dev_id = device_map[thr_id % MAX_GPUS];
	// just assign the device pointer allocated in main loop
	hipMemcpyToSymbol(HIP_SYMBOL(DState), &d_matrix, sizeof(uint64_t*), 0, hipMemcpyHostToDevice);
}

__host__
void lyra2_cpu_hash_32(int thr_id, uint32_t threads, uint32_t startNounce, uint64_t *d_hash, int order)
{
	int dev_id = device_map[thr_id % MAX_GPUS];

	uint32_t tpb = TPB52;

	if (cuda_arch[dev_id] > 500) tpb = TPB52;
	else if (cuda_arch[dev_id] == 500) tpb = TPB50;
	else if (cuda_arch[dev_id] >= 300) tpb = TPB30;
	else if (cuda_arch[dev_id] >= 200) tpb = TPB20;

	dim3 grid1((threads * 4 + tpb - 1) / tpb);
	dim3 block1(4, tpb >> 2);

	dim3 grid2((threads + 64 - 1) / 64);
	dim3 block2(64);

	if (cuda_arch[dev_id] < 500)
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(lyra2_gpu_hash_32_2), hipFuncCachePreferShared);

	lyra2_gpu_hash_32_1 << <grid2, block2 >> > (threads, startNounce, (uint2*)d_hash);

	lyra2_gpu_hash_32_2 << <grid1, block1, 192 * sizeof(uint2) * tpb >> > (threads, startNounce, d_hash);

	lyra2_gpu_hash_32_3 << <grid2, block2 >> > (threads, startNounce, (uint2*)d_hash);
	//MyStreamSynchronize(NULL, order, thr_id);
}
